#include "hip/hip_runtime.h"
/*
** Copyright 2014 Edward Walker
**
** Licensed under the Apache License, Version 2.0 (the "License");
** you may not use this file except in compliance with the License.
** You may obtain a copy of the License at
**
** http ://www.apache.org/licenses/LICENSE-2.0
**
** Unless required by applicable law or agreed to in writing, software
** distributed under the License is distributed on an "AS IS" BASIS,
** WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
** See the License for the specific language governing permissions and
** limitations under the License.
**
** Description: Cuda device code and launchers
** @author: Ed Walker
*/

#include "svm.h"
#include <stdexcept>
#include <iostream>
using namespace std;
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_math_constants.h"
#include "math.h"
#include "svm_device.h"
#include "cuda_reducer.h"
#include "svm_cache.h"
#include "sparse_bit_vector.h"

#define DEVICE_EPS	0

enum { LOWER_BOUND = 0, UPPER_BOUND = 1, FREE = 2 };

#if !USE_BITVECTOR_FORMAT
texture<float2, 1, hipReadModeElementType> 	d_tex_space;
#else
texture<float1, 1, hipReadModeElementType> 	d_tex_space;
#endif

#if USE_CONSTANT_INDEX
__constant__	int				*d_x;
#else
texture<int, 1, hipReadModeElementType> 		d_tex_x;
#endif

#if USE_BITVECTOR_FORMAT
texture<uint32_t, 1, hipReadModeElementType> 	d_tex_sparse_vector;
__device__ 		int				*d_bitvector_table;
__constant__	int				d_max_words;
#endif

__device__		int				d_kernel_type;	// enum { LINEAR, POLY, RBF, SIGMOID, PRECOMPUTED }; /* kernel_type */
__device__		int				d_svm_type;		// enum { C_SVC, NU_SVC, ONE_CLASS, EPSILON_SVR, NU_SVR };	/* svm_type */
__constant__	double			d_gamma;		// rbf, poly, and sigmoid kernel
__constant__	double			d_coef0;		// poly and sigmoid kernel
__constant__	int				d_degree;		// poly kernel
__constant__	int				d_l;			// original # SV

__constant__	CValue_t		*d_x_square;
__constant__	CValue_t		*d_QD;
__constant__	SChar_t			*d_y;
__constant__	double			d_Cp;
__constant__	double			d_Cn;

__device__		GradValue_t		*d_G;
__device__		GradValue_t		*d_alpha;
__device__		char			*d_alpha_status;

__device__		GradValue_t		d_delta_alpha_i;
__device__		GradValue_t		d_delta_alpha_j;

__device__		int2			d_solver; // member x and y hold the selected i and j working set indices respectively
__device__		int2			d_nu_solver; // member x and y hold the Gmaxp_idx and Gmaxn_idx indices respectively.  

hipError_t update_sparse_vector(uint32_t *dh_sparse_vector, int sparse_vector_size, int *dh_bitvector_table, int bitvector_table_size, int max_words)
{
	hipError_t err = hipSuccess;

#if USE_BITVECTOR_FORMAT
	if (dh_sparse_vector != NULL) {
		err = hipBindTexture(NULL, d_tex_sparse_vector, dh_sparse_vector, sparse_vector_size);
		if (err != hipSuccess) {
			fprintf(stderr, "Error binding to texture d_tex_sparse_vector\n");
			return err;
		}
	}

#if USE_SPARSE_BITVECTOR_FORMAT
	if (dh_bitvector_table == NULL) {
		fprintf(stderr, "Error: dh_bitvector_table cannot be NULL\n");
		return hipErrorInvalidConfiguration;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_bitvector_table), &dh_bitvector_table, sizeof(dh_bitvector_table));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying symbol to d_bitvector_table\n");
		return err;
	}

#endif

	if (max_words > 0) {
		err = hipMemcpyToSymbol(HIP_SYMBOL(d_max_words), &max_words, sizeof(max_words));
		if (err != hipSuccess) {
			fprintf(stderr, "Error copying to symbol d_max_words\n");
			return err;
		}
	}
#endif
	return err;
}

hipError_t update_param_constants(const svm_parameter &param, int *dh_x, cuda_svm_node *dh_space, size_t dh_space_size, int l)
{
	hipError_t err;
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_l), &l, sizeof(l));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_l\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_type), &param.kernel_type, sizeof(param.kernel_type));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_kernel_type\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_svm_type), &param.svm_type, sizeof(param.svm_type));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_svm_type\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_gamma), &param.gamma, sizeof(param.gamma));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_gamma\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_coef0), &param.coef0, sizeof(param.coef0));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_coef0\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_degree), &param.degree, sizeof(param.degree));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_degree\n");
		return err;
	}

#if USE_CONSTANT_INDEX
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_x), &dh_x, sizeof(dh_x));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_x\n");
		return err;
	}
#else
	err = hipBindTexture(0, d_tex_x, dh_x, l*sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "Error binding to d_tex_space\n");
		return err;
	}
#endif

	err = hipBindTexture(0, d_tex_space, dh_space, dh_space_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Error binding to d_tex_space\n");
		return err;
	}

	return err;
}

hipError_t update_solver_variables(SChar_t *dh_y, CValue_t *dh_QD, GradValue_t *dh_G, GradValue_t *dh_alpha, char *dh_alpha_status, double Cp, double Cn)
{
	hipError_t err;

	err = hipMemcpyToSymbol(HIP_SYMBOL(d_y), &dh_y, sizeof(dh_y));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_y\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_QD), &dh_QD, sizeof(dh_QD));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_QD\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_G), &dh_G, sizeof(dh_G));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_G\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &dh_alpha, sizeof(dh_alpha));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_alpha\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_alpha_status), &dh_alpha_status, sizeof(dh_alpha_status));
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying to symbol d_alpha_status\n");
		return err;
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(d_Cp), &Cp, sizeof(Cp));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_Cp\n");
		return err;
	}
	err = hipMemcpyToSymbol(HIP_SYMBOL(d_Cn), &Cn, sizeof(Cn));
	if (err != hipSuccess) {
		fprintf(stderr, "Error with copying to symbol d_Cn\n");
		return err;
	}
	return err;
}

hipError_t update_rbf_variables(CValue_t *dh_x_square)
{
	hipError_t err;
	if (dh_x_square != NULL) {
		err = hipMemcpyToSymbol(HIP_SYMBOL(d_x_square), &dh_x_square, sizeof(dh_x_square));
		if (err != hipSuccess) {
			fprintf(stderr, "Error copying to symbol d_x_square\n");
			return err;
		}
	}
	return err;
}

void unbind_texture()
{
	hipUnbindTexture(d_tex_space);

#if USE_BITVECTOR_FORMAT
	hipUnbindTexture(d_tex_sparse_vector);
#endif

#if !USE_CONSTANT_INDEX
	hipUnbindTexture(d_tex_x);
#endif
}

__device__ __forceinline__ 
cuda_svm_node get_col_value(int i)
{
	return tex1Dfetch(d_tex_space, i);
}

__device__ __forceinline__ 
int get_x(int i)
{
#if USE_CONSTANT_INDEX
	return d_x[i];
#else
	return tex1Dfetch(d_tex_x, i);
#endif
}

#if !USE_BITVECTOR_FORMAT
/**
Compute dot product of 2 vectors
*/
__device__ 
CValue_t dot(int i, int j)
{
	int i_col = get_x(i);
	int j_col = get_x(j);
	/**
	remember: 
	cuda_svm_node.y == svm_node.index
	cuda_svm_node.x == svm_node.value
	*/
	cuda_svm_node x = get_col_value(i_col);
	cuda_svm_node y = get_col_value(j_col);

	double sum = 0;
	while (x.y != -1 && y.y != -1)
	{
		if (x.y == y.y)
		{
			sum += x.x * y.x;
			x = get_col_value(++i_col);
			y = get_col_value(++j_col);
		}
		else
		{
			if (x.y > y.y) {
				y = get_col_value(++j_col);
			}
			else {
				x = get_col_value(++i_col);
			}
		}
	}
	return sum;
}
#else

__device__ __forceinline__ 
uint32_t get_bitvector(int i)
{
	return tex1Dfetch(d_tex_sparse_vector, i);
}

#if USE_SPARSE_BITVECTOR_FORMAT
__device__ __forceinline__
int get_bitvector_table(int i)
{
	return d_bitvector_table[i];
}

__device__ __forceinline__
int get_next_idx(int idx, size_t &run, uint32_t &pattern, int &poffset)
{
#if BITVECTOR_16BIT
	size_t sizeof_run = 2;
#else
	size_t sizeof_run = 4;
#endif
	if ((pattern & BIT_MASK) == 0)
		return -1;

	bool done;
	do {
		idx += (pattern & MAX_RUN);
		done = (pattern & BIT_SET);
		pattern >>= SHIFT_BITS;
		++run;
		if (run == sizeof_run) {
			pattern = get_bitvector(poffset++);
			run = 0;
		}
	} while (!done);
	return idx;
}

/**
  Compute dot product of 2 vectors
  */
__device__ 
CValue_t dot(int i, int j)
{
	int i_off = get_x(i);
	int j_off = get_x(j);

	int i_poffset = get_bitvector_table(i);
	int j_poffset = get_bitvector_table(j);

	uint32_t i_pattern, j_pattern;
	int i_idx = 0, j_idx = 0;
	size_t i_run = 0, j_run = 0;

	i_pattern = get_bitvector(i_poffset++); // fetch the index mask for i
	i_idx = get_next_idx(i_idx, i_run, i_pattern, i_poffset);

	j_pattern = get_bitvector(j_poffset++); // fetch the index mask for j
	j_idx = get_next_idx(j_idx, j_run, j_pattern, j_poffset);

	CValue_t sum = 0;
	while (i_idx != -1 && j_idx != -1) {
		if (i_idx == j_idx) {
			cuda_svm_node x = get_col_value(i_off++);
			cuda_svm_node y = get_col_value(j_off++);
			sum += x.x * y.x;
			i_idx = get_next_idx(i_idx, i_run, i_pattern, i_poffset);
			j_idx = get_next_idx(j_idx, j_run, j_pattern, j_poffset);
		}
		else if (i_idx < j_idx) {
			i_off++;
			i_idx = get_next_idx(i_idx, i_run, i_pattern, i_poffset);
		}
		else {
			j_off++;
			j_idx = get_next_idx(j_idx, j_run, j_pattern, j_poffset);
		}
	}
	return sum;
}

#else
__device__ __forceinline__ 
uint32_t least_significant_bit(uint32_t x, uint32_t &x_nobit)
{
	x_nobit = x & (x - 1);
	return x & ~x_nobit;
}

/**
  Compute dot product of 2 vectors
  */
__device__ 
CValue_t dot(int i, int j)
{
	int i_off = get_x(i);
	int j_off = get_x(j);
	size_t i_poffset = i * d_max_words;
	size_t j_poffset = j * d_max_words;
	/**
		remember:
		cuda_svm_node.x == svm_node.value
	*/
	uint32_t x_pattern, y_pattern;
	CValue_t sum = 0;
	for (int k1 = 0; k1 < d_max_words; k1++) {
		x_pattern = get_bitvector(i_poffset++); // fetch the index mask for i
		y_pattern = get_bitvector(j_poffset++); // fetch the index mask for j

		if (x_pattern == 0 && y_pattern == 0)
			continue;

		uint32_t bx = 0, by = 0;
		uint32_t xbit = 0, ybit = 0;

		if (x_pattern > 0) {
			xbit = least_significant_bit(x_pattern, bx); // get the first least significant bit in x
		}
		if (y_pattern > 0) {
			ybit = least_significant_bit(y_pattern, by); // get the first least significant bit in y
		}

		do {
			bool move_x = false, move_y = false;
			if (xbit == ybit) { // both bits are in the same position
				// index matches! hence we multiply
				cuda_svm_node x = get_col_value(i_off);
				cuda_svm_node y = get_col_value(j_off);
				sum += x.x * y.x;

				move_x = true;
				move_y = true;
			}
			else if (y_pattern == 0) {
				move_x = true;
			}
			else if (x_pattern == 0) {
				move_y = true;
			}
			else if (ybit < xbit) {
				move_y = true;
			}
			else if (xbit < ybit) {
				move_x = true;
			}

			if (move_x) {
				i_off++;
				x_pattern = bx;
				xbit = least_significant_bit(x_pattern, bx); // move to the next bit in x
			}
			if (move_y) {
				j_off++;
				y_pattern = by;
				ybit = least_significant_bit(y_pattern, by); // move to the next bit in y
			}
		} while (x_pattern > 0 || y_pattern > 0);
	}
	return sum;
}
#endif
#endif

__device__ 
CValue_t device_kernel_rbf(const int &i, const int &j)
{
	CValue_t q = d_x_square[i] + d_x_square[j] - 2 * dot(i, j);
	return exp(-(CValue_t)d_gamma * q);
}

__device__ 
CValue_t device_kernel_poly(const int &i, const int &j)
{
	return pow((CValue_t)d_gamma * dot(i, j) + (CValue_t)d_coef0, d_degree);
}

__device__ 
CValue_t device_kernel_sigmoid(const int &i, const int &j)
{
	return tanh((CValue_t)d_gamma * dot(i, j) + (CValue_t)d_coef0);
}

__device__ 
CValue_t device_kernel_linear(const int &i, const int &j)
{
	return dot(i, j);
}

__device__ 
CValue_t device_kernel_precomputed(const int &i, const int &j)
{
	int i_col = get_x(i);
	int j_col = get_x(j);
	int offset = static_cast<int>(get_col_value(j_col).x);
	return get_col_value(i_col + offset).x;
	// return x[i][(int)(x[j][0].value)].value;
}

/**
Returns the product of the kernel function multiplied with rc
@param i	index i
@param j	index j
@param rc	multiplier for the kernel function
*/
__device__ __forceinline__ 
CValue_t kernel(const int &i, const int &j, const CValue_t &rc)
{
	switch (d_kernel_type)
	{
	case RBF:
		return rc * device_kernel_rbf(i, j);
	case POLY:
		return rc * device_kernel_poly(i, j);
	case LINEAR:
		return rc * device_kernel_linear(i, j);
	case SIGMOID:
		return rc * device_kernel_sigmoid(i, j);
	case PRECOMPUTED:
		return rc * device_kernel_precomputed(i, j);
	}

	return 0;
}

/**
	Implements schar *SVR_Q::sign
	[0..l-1] --> 1
	[l..2*l) --> -1
*/
__device__ __forceinline__ 
SChar_t device_SVR_sign(int i)
{
	return (i < d_l ? 1 : -1);
}

/**
	Implements int *SVR_Q::index
	[0..l-1] --> [0..l-1]
	[l..2*l) --> [0..1-1]
*/
__device__ __forceinline__ 
int device_SVR_real_index(int i)
{
	return (i < d_l ? i : (i - d_l));
}

__device__ 
CValue_t cuda_evalQ(int i, int j)
{
	CValue_t rc = 1;

	switch (d_svm_type)
	{
	case C_SVC:
	case NU_SVC:
		// SVC_Q
		rc = (CValue_t)(d_y[i] * d_y[j]);
		break;
	case ONE_CLASS:
		// ONE_CLASS_Q - nothing to do
		break;
	case EPSILON_SVR:
	case NU_SVR:
		// SVR_Q
		rc = (CValue_t)(device_SVR_sign(i) * device_SVR_sign(j));
		i = device_SVR_real_index(i); // use the kernel calculation
		j = device_SVR_real_index(j); // use for kernel calculation
		break;
	}

	return kernel(i, j, rc);
}

__global__ 
void cuda_find_min_idx(CValue_t *obj_diff_array, int *obj_diff_indx, CValue_t *result_obj_min, int *result_indx, int N)
{
	D_MinIdxReducer func(obj_diff_array, obj_diff_indx, result_obj_min, result_indx); // Class defined in CudaReducer.h
	device_block_reducer(func, N); // Template function defined in CudaReducer.h
	if (blockIdx.x == 0)
		d_solver.y = func.return_idx();
}

__device__ 
void device_compute_obj_diff(int i, int j, CValue_t Qij, GradValue_t Gmax, CValue_t *dh_obj_diff_array, int *result_indx)
{

	dh_obj_diff_array[j] = CVALUE_MAX;
	result_indx[j] = -1;
	if (d_y[j] == 1)
	{
		if (!(d_alpha_status[j] == LOWER_BOUND)/*is_lower_bound(j)*/)
		{
			GradValue_t grad_diff = Gmax + d_G[j];
			if (grad_diff > DEVICE_EPS) // original: grad_diff > 0
			{
				CValue_t quad_coef = d_QD[i] + d_QD[j] - 2.0 * d_y[i] * Qij;
				CValue_t obj_diff = CVALUE_MAX;

				if (quad_coef > 0) {
					obj_diff = -(grad_diff*grad_diff) / quad_coef;
				}
				else {
					obj_diff = -(grad_diff*grad_diff) / TAU;
				}
				CHECK_FLT_RANGE(obj_diff);
				CHECK_FLT_INF(obj_diff);
				dh_obj_diff_array[j] = obj_diff;
				result_indx[j] = j;
			}

		}
	}
	else
	{
		if (!(d_alpha_status[j] == UPPER_BOUND) /*is_upper_bound(j)*/)
		{
			GradValue_t grad_diff = Gmax - d_G[j];
			if (grad_diff > DEVICE_EPS) // original: grad_diff > 0
			{
				CValue_t quad_coef = d_QD[i] + d_QD[j] + 2.0 * d_y[i] * Qij;
				CValue_t obj_diff = CVALUE_MAX;

				if (quad_coef > 0) {
					obj_diff = -(grad_diff*grad_diff) / quad_coef;
				}
				else {
					obj_diff = -(grad_diff*grad_diff) / TAU;
				}
				CHECK_FLT_RANGE(obj_diff);
				CHECK_FLT_INF(obj_diff);
				dh_obj_diff_array[j] = obj_diff;
				result_indx[j] = j;
			}
		}
	}

}

__global__ 
void cuda_compute_obj_diff(GradValue_t Gmax, CValue_t *dh_obj_diff_array, int *result_indx, int N)
{
	int i = d_solver.x;

	for (int j = blockDim.x * blockIdx.x + threadIdx.x;
		j < N;
		j += blockDim.x * gridDim.x) {	

		CValue_t Qij;
		bool valid;
		CValue_t *Qi = cache_get_Q(i, valid, STAGE_AREA_I); // staged for later use and update
		if (valid) { // reuse what we already have
			Qij = Qi[j];
		}
		else {
			Qij = cuda_evalQ(i, j);
			Qi[j] = Qij;
		}

		device_compute_obj_diff(i, j, Qij, Gmax, dh_obj_diff_array, result_indx);
	}
}

__global__ 
void cuda_compute_obj_diff_SVR(GradValue_t Gmax, CValue_t *dh_obj_diff_array, int *result_indx, int N)
{
	int i = d_solver.x;

	for (int j = blockDim.x * blockIdx.x + threadIdx.x;
		j < N;
		j += blockDim.x * gridDim.x) {

		CValue_t Qij1, Qij2;
		bool valid;
		CValue_t *Qi = cache_get_Q(i, valid, STAGE_AREA_I); // staged for later use and update
		if (valid) { // reuse what we already have
			Qij1 = Qi[j];
			Qij2 = Qi[j + d_l];
		}
		else {
			Qij1 = cuda_evalQ(i, j);
			Qi[j] = Qij1;

			Qij2 = -Qij1;
			Qi[j + d_l] = Qij2;
		}

		device_compute_obj_diff(i, j, Qij1, Gmax, dh_obj_diff_array, result_indx);
		device_compute_obj_diff(i, j + d_l, Qij2, Gmax, dh_obj_diff_array, result_indx);
	}
}

__global__ 
void cuda_update_gradient(int N)
{
	int i = d_solver.x; // selected i index
	int j = d_solver.y; // selected j index

	for (int k = blockIdx.x * blockDim.x + threadIdx.x; 
		k < N;
		k += blockDim.x * gridDim.x) {

		CValue_t *Qi, *Qj;
		CValue_t Qik, Qjk;

		Qi = cache_get_Stage(i, STAGE_AREA_I);
		if (Qi) {
			Qik = Qi[k];
		}
		else {
			Qik = cuda_evalQ(i, k);
		}

		bool valid;
		Qj = cache_get_Q(j, valid, STAGE_AREA_J);
		if (valid) {
			Qjk = Qj[k];
		}
		else {
			Qjk = cuda_evalQ(j, k);
			Qj[k] = Qjk;
		}

		d_G[k] += (Qik* d_delta_alpha_i + Qjk * d_delta_alpha_j);
	}
}

__global__ 
void cuda_update_gradient_SVR(int N)
{
	int i = d_solver.x; // selected i index
	int j = d_solver.y; // selected j index

	for (int k = blockIdx.x * blockDim.x + threadIdx.x; 
		k < N;
		k += blockDim.x * gridDim.x) {

		CValue_t *Qi, *Qj;
		CValue_t Qik1, Qik2, Qjk1, Qjk2;

		Qi = cache_get_Stage(i, STAGE_AREA_I);
		if (Qi) {
			Qik1 = Qi[k];
			Qik2 = Qi[k + d_l];
		} else {
			Qik1 = cuda_evalQ(i, k);
			Qik2 = cuda_evalQ(i, k + d_l);
		}

		bool valid;
		Qj = cache_get_Q(j, valid, STAGE_AREA_J);
		if (valid) {
			Qjk1 = Qj[k];
			Qjk2 = Qj[k + d_l];
		}
		else {
			Qjk1 = cuda_evalQ(j, k);
			Qj[k] = Qjk1;

			Qjk2 = -Qjk1;
			Qj[k + d_l] = Qjk2;
		}

		d_G[k] += (Qik1 * d_delta_alpha_i + Qjk1 * d_delta_alpha_j);
		d_G[k + d_l] += (Qik2 * d_delta_alpha_i + Qjk2 * d_delta_alpha_j);
	}
}

__global__ 
void cuda_init_gradient(int start, int step, int N)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= N)
		return;

	GradValue_t acc = 0;
	for (int i = start; i < N && i < start + step; ++i)
	{
		if (!(d_alpha_status[i] == LOWER_BOUND) /*is_lower_bound(i)*/)
		{
			acc += d_alpha[i] * cuda_evalQ(i, j);
		}
	}

	d_G[j] += acc;
}

#if USE_DOUBLE_GRADIENT // needed if we are storing double gradient values
/**
double version of atomicAdd
*/
__device__ 
double atomicAdd(double * address, double val)
{
	unsigned long long int *address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}
#endif

__device__ 
GradValue_t device_compute_gradient(int i, int j)
{
	if (!(d_alpha_status[i] == LOWER_BOUND)) /* !is_lower_bound(i) */
	{
		return d_alpha[i] * cuda_evalQ(i, j);
	}
	else
		return 0;
}


__device__ __forceinline__ 
GradValue_t warpReduceSum(GradValue_t val) 
{
#if __CUDA_ARCH__ >= 300
	for (int offset = warpSize/2; offset > 0; offset /= 2) {
		val += __shfl_down(val, offset);
	}
#endif
	return val;
}

__device__ __forceinline__ 
GradValue_t blockReduceSum(GradValue_t val) 
{
#if __CUDA_ARCH__ >= 300
	static __shared__ GradValue_t shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val);     // Each warp performs partial reduction

	if (lane==0) shared[wid]=val;	// Write reduced value to shared memory

	__syncthreads();              // Wait for all partial reductions

	//read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid==0) val = warpReduceSum(val); //Final reduce within first warp
#endif
	return val;
}

__global__ 
void cuda_init_gradient_block2(int startj, int N)
{
	int j = blockIdx.y * blockDim.y + threadIdx.y + startj;
	if (j >= N)
		return ;

	GradValue_t sum = 0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
			i < N;
			i += blockDim.x * gridDim.x) {
		sum += device_compute_gradient(i, j);
	}

#if BLOCK_ATOMIC_REDUCE
	sum = blockReduceSum(sum);

	if (threadIdx.x == 0) { 
		atomicAdd(&d_G[j], sum);
	}
#else
	sum = warpReduceSum(sum);

	if (threadIdx.x & (warpSize - 1) == 0) { 
		atomicAdd(&d_G[j], sum);
	}
#endif

	return;
}

__global__ 
void cuda_init_gradient_block1(int startj, int N)
{
	int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y + startj;

	if (j >= N || i >= N)
		return;

	D_GradientAdder func(j, N);
	device_block_reducer(func, N);

	if (threadIdx.x == 0) { // every block in the x-axis (ie. i)
		GradValue_t s = func.return_sum();
		atomicAdd(&d_G[j], s);
	}

	return;
}

/**
	Initializes the gradient vector on the device
	@param block_size	number of threads per block
	@param startj		starting index j for G_j
	@param stepj		number of steps from startj to update
	@param N			size of gradient vector
*/
void init_device_gradient2(int block_size, int startj, int stepj, int N)
{
	dim3 grid;
	// the number of blocks in the ith dimension
	grid.x = std::min((N + block_size-1) / block_size, 1024);
	// the number of blocks in the jth dimension == G_j that will be updated
	grid.y = stepj; 

	dim3 block;
	block.x = block_size; // number of threads in the ith dimension
	block.y = 1; // number of threads per block in the jth dimension (one thread per block)
	cuda_init_gradient_block2 << <grid, block >> > (startj, N);
	check_cuda_kernel_launch("fail in cuda_init_gradient_block2");
}

/**
	Initializes the gradient vector on the device
	@param block_size	number of threads per block
	@param startj		starting index j for G_j
	@param stepj		number of steps from startj to update
	@param N			size of gradient vector
*/
void init_device_gradient1(int block_size, int startj, int stepj, int N)
{
	int reduce_block_size = 2 * block_size;
	dim3 grid;
	// the number of blocks in the ith dimension
	grid.x = (N+reduce_block_size-1) / reduce_block_size;
	// the number of blocks in the jth dimension == G_j that will be updated
	grid.y = stepj; 

	dim3 block;
	block.x = block_size; // number of threads in the ith dimension
	block.y = 1; // number of threads per block in the jth dimension (one thread per block)
	
	size_t shared_mem = block.x * sizeof(GradValue_t);
	cuda_init_gradient_block1 << <grid, block, shared_mem >> > (startj, N);
	check_cuda_kernel_launch("fail in cuda_init_gradient_block1");
}

__global__ 
void cuda_find_gmax(find_gmax_param param, int N, bool debug)
{
	D_GmaxReducer func(param.dh_gmax, param.dh_gmax2, param.dh_gmax_idx, param.result_gmax, 
		param.result_gmax2, param.result_gmax_idx, debug); // class defined in CudaReducer.h

	device_block_reducer(func, N); // Template function defined in CudaReducer.h

	if (blockIdx.x == 0 && threadIdx.x == 0)
		d_solver.x = func.return_idx();
}

__global__ 
void cuda_setup_x_square(int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	d_x_square[i] = dot(i, i);
}

__global__ 
void cuda_setup_QD(int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N)
		return;

	d_QD[i] = kernel(i, i, 1);

	if (d_svm_type == NU_SVR || d_svm_type == EPSILON_SVR)
		d_QD[i + d_l] = d_QD[i];
}

__global__ 
void cuda_prep_gmax(GradValue_t *dh_gmax, GradValue_t *dh_gmax2, int *dh_gmax_idx, int N)
{
	int t = blockIdx.x * blockDim.x + threadIdx.x;

	if (t >= N)
		return;

	dh_gmax[t] = -GRADVALUE_MAX;
	dh_gmax2[t] = -GRADVALUE_MAX;
	dh_gmax_idx[t] = -1;
	if (d_y[t] == +1)
	{
		if (!(d_alpha_status[t] == UPPER_BOUND) /*is_upper_bound(t)*/) {
			dh_gmax[t] = -d_G[t];
			dh_gmax_idx[t] = t;
		}
		if (!(d_alpha_status[t] == LOWER_BOUND) /*is_lower_bound(t)*/) {
			dh_gmax2[t] = d_G[t];
		}
	}
	else
	{
		if (!(d_alpha_status[t] == LOWER_BOUND) /*is_lower_bound(t)*/) {
			dh_gmax[t] = d_G[t];
			dh_gmax_idx[t] = t;
		}
		if (!(d_alpha_status[t] == UPPER_BOUND) /*is_upper_bound(t)*/) {
			dh_gmax2[t] = -d_G[t];
		}
	}
}

__device__	__forceinline__ 
double device_get_C(int i)
{
	return (d_y[i] > 0) ? d_Cp : d_Cn;
}

__global__ 
void cuda_compute_alpha()
{
	int i = d_solver.x; // d_selected_i;
	int j = d_solver.y; // d_selected_j;

	GradValue_t C_i = device_get_C(i);
	GradValue_t C_j = device_get_C(j);

	GradValue_t old_alpha_i = d_alpha[i];
	GradValue_t old_alpha_j = d_alpha[j];

	if (d_y[i] != d_y[j])
	{
		GradValue_t quad_coef = d_QD[i] + d_QD[j] + 2 * cuda_evalQ(i, j); //  Q_i[j];
		if (quad_coef <= 0)
			quad_coef = TAU;
		GradValue_t delta = (-d_G[i] - d_G[j]) / quad_coef;
		GradValue_t diff = d_alpha[i] - d_alpha[j];
		d_alpha[i] += delta;
		d_alpha[j] += delta;

		if (diff > 0)
		{
			if (d_alpha[j] < 0)
			{
				d_alpha[j] = 0;
				d_alpha[i] = diff;
			}
		}
		else
		{
			if (d_alpha[i] < 0)
			{
				d_alpha[i] = 0;
				d_alpha[j] = -diff;
			}
		}
		if (diff > C_i - C_j)
		{
			if (d_alpha[i] > C_i)
			{
				d_alpha[i] = C_i;
				d_alpha[j] = C_i - diff;
			}
		}
		else
		{
			if (d_alpha[j] > C_j)
			{
				d_alpha[j] = C_j;
				d_alpha[i] = C_j + diff;
			}
		}
	}
	else
	{
		GradValue_t quad_coef = d_QD[i] + d_QD[j] - 2 * cuda_evalQ(i, j); // Q_i[j];
		if (quad_coef <= 0)
			quad_coef = TAU;
		GradValue_t delta = (d_G[i] - d_G[j]) / quad_coef;
		GradValue_t sum = d_alpha[i] + d_alpha[j];
		d_alpha[i] -= delta;
		d_alpha[j] += delta;

		if (sum > C_i)
		{
			if (d_alpha[i] > C_i)
			{
				d_alpha[i] = C_i;
				d_alpha[j] = sum - C_i;
			}
		}
		else
		{
			if (d_alpha[j] < 0)
			{
				d_alpha[j] = 0;
				d_alpha[i] = sum;
			}
		}
		if (sum > C_j)
		{
			if (d_alpha[j] > C_j)
			{
				d_alpha[j] = C_j;
				d_alpha[i] = sum - C_j;
			}
		}
		else
		{
			if (d_alpha[i] < 0)
			{
				d_alpha[i] = 0;
				d_alpha[j] = sum;
			}
		}
	}
	d_delta_alpha_i = d_alpha[i] - old_alpha_i;
	d_delta_alpha_j = d_alpha[j] - old_alpha_j;
}

__device__ 
void device_update_alpha_status(int i)
{
	if (d_alpha[i] >= device_get_C(i))
		d_alpha_status[i] = UPPER_BOUND;
	else if (d_alpha[i] <= 0)
		d_alpha_status[i] = LOWER_BOUND;
	else
		d_alpha_status[i] = FREE;
}

__global__ 
void cuda_update_alpha_status()
{
	int i = d_solver.x;
	int j = d_solver.y;

	device_update_alpha_status(i);
	device_update_alpha_status(j);

	cache_commit_Stages(i, j);
}

/*********** NU Solver ************/


__global__ 
void cuda_prep_nu_gmax(GradValue_t *dh_gmaxp, GradValue_t *dh_gmaxn, GradValue_t *dh_gmaxp2, GradValue_t *dh_gmaxn2,
	int *dh_gmaxp_idx, int *dh_gmaxn_idx, int N)
{
	int t = blockIdx.x * blockDim.x + threadIdx.x;

	if (t >= N)
		return;

	dh_gmaxp[t] = -GRADVALUE_MAX;
	dh_gmaxp2[t] = -GRADVALUE_MAX;
	dh_gmaxn[t] = -GRADVALUE_MAX;
	dh_gmaxn2[t] = -GRADVALUE_MAX;
	dh_gmaxp_idx[t] = -1;
	dh_gmaxn_idx[t] = -1;

	if (d_y[t] == +1)
	{
		if (!(d_alpha_status[t] == UPPER_BOUND) /*is_upper_bound(t)*/) {
			dh_gmaxp[t] = -d_G[t];
			dh_gmaxp_idx[t] = t;
		}
		if (!(d_alpha_status[t] == LOWER_BOUND) /*is_lower_bound(t)*/) {
			dh_gmaxp2[t] = d_G[t];
		}
	}
	else
	{
		if (!(d_alpha_status[t] == LOWER_BOUND) /*is_lower_bound(t)*/) {
			dh_gmaxn[t] = d_G[t];
			dh_gmaxn_idx[t] = t;
		}
		if (!(d_alpha_status[t] == UPPER_BOUND) /*is_upper_bound(t)*/) {
			dh_gmaxn2[t] = -d_G[t];
		}
	}
}

__device__ 
void device_compute_nu_obj_diff(int ip, int in, int j, CValue_t Qipj, GradValue_t Gmaxp, GradValue_t Gmaxn, CValue_t *dh_obj_diff_array, int *result_idx)
{

	dh_obj_diff_array[j] = CVALUE_MAX;
	result_idx[j] = -1;
	if (d_y[j] == 1)
	{
		if (!(d_alpha_status[j] == LOWER_BOUND)/*is_lower_bound(j)*/)
		{
			GradValue_t grad_diff = Gmaxp + d_G[j];
			if (grad_diff > DEVICE_EPS) // original: grad_diff > 0
			{
				CValue_t quad_coef = d_QD[ip] + d_QD[j] - 2.0 * Qipj;
				CValue_t obj_diff = CVALUE_MAX;

				if (quad_coef > 0) {
					obj_diff = -(grad_diff*grad_diff) / quad_coef;
				}
				else {
					obj_diff = -(grad_diff*grad_diff) / TAU;
				}
				CHECK_FLT_RANGE(obj_diff);
				CHECK_FLT_INF(obj_diff);
				dh_obj_diff_array[j] = obj_diff;
				result_idx[j] = j;
			}

		}
	}
	else
	{
		if (!(d_alpha_status[j] == UPPER_BOUND) /*is_upper_bound(j)*/)
		{
			GradValue_t grad_diff = Gmaxn - d_G[j];
			if (grad_diff > DEVICE_EPS) // original: grad_diff > 0
			{
				CValue_t quad_coef = d_QD[in] + d_QD[j] - 2.0 * cuda_evalQ(in, j);
				CValue_t obj_diff = CVALUE_MAX;

				if (quad_coef > 0) {
					obj_diff = -(grad_diff*grad_diff) / quad_coef;
				}
				else {
					obj_diff = -(grad_diff*grad_diff) / TAU;
				}
				CHECK_FLT_RANGE(obj_diff);
				CHECK_FLT_INF(obj_diff);
				dh_obj_diff_array[j] = obj_diff;
				result_idx[j] = j;
			}
		}
	}

}

__global__ 
void cuda_compute_nu_obj_diff(GradValue_t Gmaxp, GradValue_t Gmaxn, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	int ip = d_nu_solver.x;
	int in = d_nu_solver.y;

	for (int j = blockDim.x * blockIdx.x + threadIdx.x;
		j < N;
		j += blockDim.x * gridDim.x) {

		CValue_t Qipj;
		bool valid;
		CValue_t *Qip = cache_get_Q(ip, valid, STAGE_AREA_I); // staged for later use and update
		if (valid) { // reuse what we already have
			Qipj = Qip[j];
		}
		else {
			Qipj = cuda_evalQ(ip, j);
			Qip[j] = Qipj;
		}

		device_compute_nu_obj_diff(ip, in, j, Qipj, Gmaxp, Gmaxn, dh_obj_diff_array, result_idx);
	}
}

__global__ 
void cuda_compute_nu_obj_diff_SVR(GradValue_t Gmaxp, GradValue_t Gmaxn, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	int ip = d_nu_solver.x;
	int in = d_nu_solver.y;

	for (int j = blockDim.x * blockIdx.x + threadIdx.x;
		j < N;
		j += blockDim.x * gridDim.x) {

		CValue_t Qipj1, Qipj2;
		bool valid;
		CValue_t *Qip = cache_get_Q(ip, valid, STAGE_AREA_I); // staged for later use and update
		if (valid) { // reuse what we already have
			Qipj1 = Qip[j];
			Qipj2 = Qip[j + d_l];
		}
		else {
			Qipj1 = cuda_evalQ(ip, j);
			Qip[j] = Qipj1;

			Qipj2 = -Qipj1;
			Qip[j + d_l] = Qipj2;
		}


		device_compute_nu_obj_diff(ip, in, j, Qipj1, Gmaxp, Gmaxn, dh_obj_diff_array, result_idx);
		device_compute_nu_obj_diff(ip, in, j + d_l, Qipj2, Gmaxp, Gmaxn, dh_obj_diff_array, result_idx);
	}
}


__global__ 
void cuda_find_nu_gmax(find_nu_gmax_param param, int N)
{
	D_NuGmaxReducer func(param.dh_gmaxp, param.dh_gmaxn, param.dh_gmaxp2, param.dh_gmaxn2, param.dh_gmaxp_idx, param.dh_gmaxn_idx,
		param.result_gmaxp, param.result_gmaxn, param.result_gmaxp2, param.result_gmaxn2, param.result_gmaxp_idx, param.result_gmaxn_idx);

	device_block_reducer(func, N);

	if (blockIdx.x == 0 && threadIdx.x == 0) {
		int ip, in;
		func.return_idx(ip, in);
		d_nu_solver.x = ip;
		d_nu_solver.y = in;
	}
}



__global__ 
void cuda_find_nu_min_idx(CValue_t *obj_diff_array, int *obj_diff_idx, CValue_t *result_obj_min, int *result_idx, int N)
{
	D_MinIdxReducer func(obj_diff_array, obj_diff_idx, result_obj_min, result_idx); // Class defined in CudaReducer.h
	device_block_reducer(func, N); // Template function defined in CudaReducer.h
	if (blockIdx.x == 0) {
		int j = func.return_idx();
		d_solver.y = j; /* Gmin_idx */
		if (d_y[j] == +1)
			d_solver.x = d_nu_solver.x; /* Gmaxp_idx */
		else
			d_solver.x = d_nu_solver.y; /* Gmaxn_idx */
	}
}

/************DEVICE KERNEL LAUNCHERS***************/
void launch_cuda_setup_x_square(size_t num_blocks, size_t block_size, int N)
{
	cuda_setup_x_square << <num_blocks, block_size >> >(N);
}

void launch_cuda_setup_QD(size_t num_blocks, size_t block_size, int N)
{
	cuda_setup_QD << <num_blocks, block_size >> >(N);
}


void launch_cuda_compute_obj_diff(size_t num_blocks, size_t block_size, GradValue_t Gmax, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	cuda_compute_obj_diff << <num_blocks, block_size >> > (Gmax, dh_obj_diff_array, result_idx, N);
}

void launch_cuda_compute_obj_diff_SVR(size_t num_blocks, size_t block_size, GradValue_t Gmax, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	cuda_compute_obj_diff_SVR << <num_blocks, block_size >> > (Gmax, dh_obj_diff_array, result_idx, N);
}

void launch_cuda_update_gradient(size_t num_blocks, size_t block_size, int N)
{
	cuda_update_gradient << <num_blocks, block_size >> > (N);
}

void launch_cuda_update_gradient_SVR(size_t num_blocks, size_t block_size, int N)
{
	cuda_update_gradient_SVR << <num_blocks, block_size >> > (N);
}

void launch_cuda_init_gradient(size_t num_blocks, size_t block_size, int start, int step, int N)
{
	cuda_init_gradient << < num_blocks, block_size>> > (start, step, N);
}

void launch_cuda_prep_gmax(size_t num_blocks, size_t block_size, GradValue_t *dh_gmax, GradValue_t *dh_gmax2, int *dh_gmax_idx, int N)
{
	cuda_prep_gmax << < num_blocks, block_size>> > (dh_gmax, dh_gmax2, dh_gmax_idx, N);
}

void launch_cuda_compute_alpha(size_t num_blocks, size_t block_size)
{
	cuda_compute_alpha << <num_blocks, block_size >> >();
}

void launch_cuda_update_alpha_status(size_t num_blocks, size_t block_size)
{
	cuda_update_alpha_status << <num_blocks, block_size >> >();
}

void launch_cuda_find_min_idx(size_t num_blocks, size_t block_size, size_t share_mem_size, CValue_t *obj_diff_array, int *obj_diff_idx, CValue_t *result_obj_min, int *result_idx, int N)
{
	cuda_find_min_idx << <num_blocks, block_size, share_mem_size >> >(obj_diff_array, obj_diff_idx, result_obj_min, result_idx, N);
}

void launch_cuda_find_gmax(size_t num_blocks, size_t block_size, size_t share_mem_size, find_gmax_param param, int N, bool debug)
{
	cuda_find_gmax << <num_blocks, block_size, share_mem_size >> >(param, N, debug);
}

void launch_cuda_find_nu_min_idx(size_t num_blocks, size_t block_size, size_t share_mem_size, CValue_t *obj_diff_array, int *obj_diff_idx, CValue_t *result_obj_min, int *result_idx, int N)
{
	cuda_find_nu_min_idx << <num_blocks, block_size, share_mem_size >> >(obj_diff_array, obj_diff_idx, result_obj_min, result_idx, N);
}

void launch_cuda_find_nu_gmax(size_t num_blocks, size_t block_size, size_t share_mem_size, find_nu_gmax_param param, int N)
{
	cuda_find_nu_gmax << <num_blocks, block_size, share_mem_size >> >(param, N);
}

void launch_cuda_compute_nu_obj_diff(size_t num_blocks, size_t block_size, GradValue_t Gmaxp, GradValue_t Gmaxn, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	cuda_compute_nu_obj_diff << <num_blocks, block_size >> > (Gmaxp, Gmaxn, dh_obj_diff_array, result_idx, N);
}

void launch_cuda_compute_nu_obj_diff_SVR(size_t num_blocks, size_t block_size, GradValue_t Gmaxp, GradValue_t Gmaxn, CValue_t *dh_obj_diff_array, int *result_idx, int N)
{
	cuda_compute_nu_obj_diff_SVR << <num_blocks, block_size >> > (Gmaxp, Gmaxn, dh_obj_diff_array, result_idx, N);
}

void launch_cuda_prep_nu_gmax(size_t num_blocks, size_t block_size, GradValue_t *dh_gmaxp, GradValue_t *dh_gmaxn, GradValue_t *dh_gmaxp2, GradValue_t *dh_gmaxn2,
	int *dh_gmaxp_idx, int *dh_gmaxn_idx, int N)
{
	cuda_prep_nu_gmax << <num_blocks, block_size >> > (dh_gmaxp, dh_gmaxn, dh_gmaxp2, dh_gmaxn2, dh_gmaxp_idx, dh_gmaxn_idx, N);
}



/**************** DEBUGGING ********************/
/**
useful for peeking at various misc values when debugging
*/
__global__ 
void cuda_peek(int i, int j)
{
	printf("Q(%d,%d)=%g\n", i, j, cuda_evalQ(i, j));
}


